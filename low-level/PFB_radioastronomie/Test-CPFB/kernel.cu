
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#include "gtest/gtest.h"

int main(int argc, char** argv)
{
	cout << "******* TESTs *******" << endl;
	::testing::InitGoogleTest(&argc, argv);
	RUN_ALL_TESTS();
	cout << endl;

	return 0;
}
