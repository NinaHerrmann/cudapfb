#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <math.h>
#include <stdio.h>
#include <hipfft/hipfft.h>
#include <iostream>
#include <fstream>
#include <sstream>
#include <chrono>
#include <vector>
#include <cmath>
#include <cstring>
#include <iosfwd>

#define M_PI       3.14159265358979323846   // pi

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

// Complex numbers operations
static __device__ __host__ inline float2 CplxAdd(float2 a, float2 b) {
	float2 c; c.x = a.x + b.x; c.y = a.y + b.y; return c;
}

static __device__ __host__ inline float2 CplxInv(float2 a) {
	float2 c; c.x = -a.x; c.y = -a.y; return c;
}

static __device__ __host__ inline float2 CplxMul(float2 a, float2 b) {
	float2 c; c.x = a.x * b.x - a.y + b.y; c.y = a.x * b.y + a.y * b.x; return c;
}

/**
 * Reorders array by bit-reversing the indexes.
 */
__global__ void bitrev_reorder(float2* __restrict__ r, float2* __restrict__ d, int s, size_t nthr) {
	int id = blockIdx.x * nthr + threadIdx.x;
	//if (threadIdx.x == 0) printf("take id %d write to %d\n", id, __brev(id) >> (32 - s));
	r[__brev(id) >> (32 - s)] = d[id];
}
/**
 * Inner part of FFT loop. Contains the procedure itself.
 */
__device__ void inplace_fft_inner(float2* __restrict__ r, int j, int k, int m, int n) {
	if (j + k + m / 2 < n) {
		float2 t, u;

		t.x = __cosf((2.0 * M_PI * k) / (1.0 * m));
		t.y = -__sinf((2.0 * M_PI * k) / (1.0 * m));

		u = r[j + k];
		t = CplxMul(t, r[j + k + m / 2]);

		r[j + k] = CplxAdd(u, t);
		r[j + k + m / 2] = CplxAdd(u, CplxInv(t));
	}
}
/**
 * Middle part of FFT for small scope paralelism.
 */
__global__ void inplace_fft(float2* __restrict__ r, int j, int m, int n, size_t nthr) {
	int k = blockIdx.x * nthr + threadIdx.x;
	inplace_fft_inner(r, j, k, m, n);
}

/**
 * Outer part of FFT for large scope paralelism.
 */
__global__ void inplace_fft_outer(float2* __restrict__ r, int m, int n, size_t nthr) {
	int j = (blockIdx.x * nthr + threadIdx.x) * m;

	for (int k = 0; k < m / 2; k++) {
		inplace_fft_inner(r, j, k, m, n);
	}
}
__device__ float2 complex_mult(float2 returnvalue, float2 Ai) {
	float2 result;
	result.x = (returnvalue.x * Ai.x - returnvalue.y * Ai.y);
	result.y = (returnvalue.x * Ai.y + returnvalue.y * Ai.x);
	return result;
}
__device__ float2 complex_add(float2 returnvalue, float2 Ai) {
	float2 result;
	result.x = returnvalue.x + Ai.x;
	result.y = returnvalue.y + Ai.y;
	return result;
}
__global__ void muesli_combine(float2* T, float2* R, int log2size, int j, int Problemsize) {
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int b = i >> (log2size - j - 1);   int b2 = 0;
	for (int l = 0; l <= j; l++) {
		b2 = (b & 1) ? 2 * b2 + 1 : 2 * b2;
		b >>= 1;
	}
	float2 Ai = R[i];
	double v = 2.0 * M_PI / Problemsize * (b2 << (log2size - j - 1));
	float2 returnvalue;
	returnvalue.x = cos(v);
	returnvalue.y = sin(v);
	//inline complex combine(const DistributedArray<complex> & T, int j, int i, complex Ai) {
	float2 res;
	(i & (1 << log2size - 1 - j)) ? res = complex_add(T[i], complex_mult(returnvalue, Ai)) : res = complex_add(Ai, complex_mult(returnvalue, T[i]));
	//printf("\nWrite %f %f to %d",res.x, res.y, i);
	T[i] = res;
	//}
}

__device__ int dev_bitcomplement(int k, int i) {
	return i ^ (1 << k);
}

__global__ void muesli_fetch(float2* T, float2* R, int log2size, int j) {
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	//fetch(const DistributedArray<complex> & R, int j, int i, complex Ti) {
	//return R.get(bitcomplement(log2size - 1 - j, i));
	T[i] = R[dev_bitcomplement(log2size - 1 - j, i)];
}

__global__ void bitcomplement(float2* result, float2* temp) {
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	// TODO return i ^ (1<<k);
}


int main()
{
    const int arraySize = 5;
    const int a[arraySize] = { 1, 2, 3, 4, 5 };
    const int b[arraySize] = { 10, 20, 30, 40, 50 };
    int c[arraySize] = { 0 };
	// Copy data to GPU
	float2* r;
	float2* dn;
	float2* h_dn;
	int n = 16;
	size_t data_size = n * sizeof(float2);
	
	float2* d_Index;
	float2* d_Index_output;
	float2* d_Vier;
	float2* d_Vier_output;
	float2* h_Index;
	float2* h_vier;
	h_Index = (float2*)malloc(data_size);
	h_vier = (float2*)malloc(data_size);

	hipMalloc((void**)& d_Index, sizeof(float2) * n);
	hipMalloc((void**)& d_Index_output, sizeof(float2) * n);
	hipMalloc((void**)& d_Vier, sizeof(float2) * n);
	hipMalloc((void**)& d_Vier_output, sizeof(float2) * n);
	printf("\nInput:");
	for (int i = 0; i < n; i++) {
		h_Index[i].x = i;
		h_Index[i].y = i;
	}
	printf("\nVier:");
	for (int i = 0; i < n; i++) {
		h_vier[i].x = 4;
		h_vier[i].y = 4;
	}
	hipMemcpy(d_Index, h_Index, data_size, hipMemcpyHostToDevice);
	hipMemcpy(d_Vier, h_vier, data_size, hipMemcpyHostToDevice);
    // Add vectors in parallel.
	int threads = 16;
	int log2size = log2(n);
	dim3 dim_blocks(n / threads);
	dim3 dim_threads(threads);
	bitrev_reorder << <dim_blocks, dim_threads >> > (d_Index_output, d_Index, log2size, threads);
	bitrev_reorder << <dim_blocks, dim_threads >> > (d_Vier_output, d_Vier, log2size, threads);
	hipMemcpy(h_Index, d_Index_output, data_size, hipMemcpyDeviceToHost);
	hipMemcpy(h_vier, d_Vier_output, data_size, hipMemcpyDeviceToHost);
	printf("\nIndex: ");
	for (int i = 0; i < 16; i++) {
		if (i % 4 == 0) {
			printf("\n");
		}
		printf("(%f,%f)",h_Index[i].x,	h_Index[i].y);
	}
	printf("\nVier: ");

	for (int i = 0; i < 16; i++) {
		if (i % 4 == 0) {
			printf("\n");
		}
		printf("(%f,%f)", h_vier[i].x, h_vier[i].y);	
	}
	for (int j = 0; j < log2size; j++) {
		// T.mapIndexInPlace(curry(fetch)(R)(j));
		// (float2* T, float2* R, int log2size, int j)
		muesli_fetch << <(1), threads >> > (d_Vier, d_Vier_output, log2size, j);
		muesli_fetch << <(1), threads >> > (d_Index, d_Index_output, log2size, j);
		hipMemcpy(h_vier, d_Vier, data_size, hipMemcpyDeviceToHost);

		printf("\nFetch: Iteration %d Vier : ", j);

		for (int i = 0; i < 16; i++) {
			if (i % 4 == 0) {
				printf("\n");
			}
			printf("(%f,%f)", h_vier[i].x, h_vier[i].y);
		}
		hipMemcpy(h_Index, d_Index, data_size, hipMemcpyDeviceToHost);

		printf("\nFetch: Iteration %d Index : ", j);

		for (int i = 0; i < 16; i++) {
			if (i % 4 == 0) {
				printf("\n");
			}
			printf("(%f,%f)", h_Index[i].x, h_Index[i].y);
		}
		muesli_combine << <(1), threads >> > (d_Vier_output, d_Vier, log2size, j, 16);
		muesli_combine << <(1), threads >> > (d_Index_output, d_Index, log2size, j, 16);
		hipMemcpy(h_vier, d_Vier_output, data_size, hipMemcpyDeviceToHost);

		printf("\nCombine: Iteration %d Vier : ", j);

		for (int i = 0; i < 16; i++) {
			if (i % 4 == 0) {
				printf("\n");
			}
			printf("(%f,%f)", h_vier[i].x, h_vier[i].y);
		}
		hipMemcpy(h_Index, d_Index_output, data_size, hipMemcpyDeviceToHost);

		printf("\nCombine: Iteration %d Index : ", j);

		for (int i = 0; i < 16; i++) {
			if (i % 4 == 0) {
				printf("\n");
			}
			printf("(%f,%f)", h_Index[i].x, h_Index[i].y);
		}
		
	}
	float2* im_result;
	im_result = (float2*)malloc(data_size);
	// log2size = 9
	hipDeviceSynchronize();
	// Iterative FFT (with loop paralelism balancing)

	//r = dn;
	// move everything
	//bitcomplement<< <((float)n / m / threads), threads >> > (dn, r);
	// T.permutePartition(curry(bitcomplement)(log2p - 1 - j));
	// R.mapIndexInPlace(curry(combine)(T)(j)); --> calculate individuals
	int Problemsize = data_size;
	int j = 0;
	//muesli_combine << <((float)n / m / threads), threads >> > (dn, r, log2size, j, Problemsize);

	
	//hipFree(r);
	//hipFree(dn);
	// Making a cufft comparison...
	hipfftHandle plan;
	hipfftComplex* data;
	hipfftComplex* cufft_output;
	float2* cufft_h_dn;
	cufft_h_dn = (float2*)malloc(data_size);
	hipMalloc((void**)& data, sizeof(hipfftComplex) * 1024);
	hipMalloc((void**)& cufft_output, sizeof(hipfftComplex) * 1024);
	for (int i = 0; i < 1024; i++) {
		cufft_h_dn[i].x = 4;
		cufft_h_dn[i].y = 4;
		//printf("(%f,%f)", h_dn[i].x, h_dn[i].y);
	}

	hipMemcpy(data, cufft_h_dn, data_size, hipMemcpyHostToDevice);
	hipfftPlan1d(&plan, 32, HIPFFT_C2C, 1024);
	hipError_t cudaStatus = hipGetLastError();

	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cufft plan failed: %s\n", hipGetErrorString(cudaStatus));
	}
	hipfftExecC2C(plan, data, cufft_output, HIPFFT_FORWARD);
	
	hipDeviceSynchronize();
	hipMemcpy(cufft_h_dn, cufft_output, data_size, hipMemcpyDeviceToHost);
	for (int i = 0; i < 10; i++) {
		//(cufft_h_dn[i].x == result[i].x) ? printf("same: %f==%f;\n", cufft_h_dn[i].x, result[i].x) : printf("%f!=%f; %f!=%f; \n", cufft_h_dn[i].x, result[i].x, cufft_h_dn[i].y, result[i].y);
	}

	hipfftDestroy(plan);
	hipFree(data);

    //hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    //printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n", c[0], c[1], c[2], c[3], c[4]);

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;

}
