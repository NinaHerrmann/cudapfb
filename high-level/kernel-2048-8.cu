#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include "timer.cuh"
#include ""
#include <stdio.h>
#include <array>
#include <iostream>
#include <time.h>
#include <chrono>
#include <stdlib.h>
#include <stdint.h>
#include <stdio.h>
#include <vector>
#include <hip/hip_runtime.h>
#include <stdexcept>
#include <sstream>

// Global Constants
const size_t number_of_processes = 1;
const size_t number_of_gpus = 1;
// Global Variables
// Temporary Variables
size_t tmp_size_t = 0;
size_t fetch_size = 0;

//------ DataStructure Declarations -----
	//Declare everything for input
std::vector<float> h_input(250013696); //Size:250013696, localSize: 250013696, gpuSize: 250013696
std::vector<float> h_input_glbcopy(250013696);
float* d_input0_loccopy;
float* d_input0_glbcopy;

//Declare everything for coeff
std::vector<float> h_coeff(16384); //Size:480, localSize: 480, gpuSize: 480
std::vector<float> h_coeff_glbcopy(16384);
float* d_coeff0_loccopy;
float* d_coeff0_glbcopy;

//------ Kernel Functions -----
__global__ void print(float* y) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	printf("y at %d:%f\n", index, y[index]);
}
// h_input_xOffset + 0, 0, (ntaps), (nchans), (nspectra), d_input0, d_output0
__global__ void FIR_MapIndexInPlaceSkeleton_array(int a, int y, int taps, int rowOffset, int nspectra, float* p_input, float* p_coeff, float* d_output) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int channels = rowOffset;
	float newa = 0;
	for (int j = 0; ((j) < (taps)); j++) {
		if (((index + ((j) * (channels)))) <= (nspectra * channels)) {
			newa += (// TODO: ExpressionGenerator.generateCollectionElementRef: Array, global indices, distributed
				p_input[(index + ((j) * (channels)))]
				* // TODO: ExpressionGenerator.generateCollectionElementRef: Array, global indices, distributed
				p_coeff[(((j) * (channels)))]
				);
		}
		else {
			// Do nothing since we do not have further data
		}
	}
	d_output[index] = (newa);
}

void writefloattofilevector(std::size_t datasize, std::vector<float> data) {
	for (int n = 0; n < (datasize); n++) {
		float x = (rand() / (float)RAND_MAX);
		// printf("\n Write at place %d value %f", n , x);
		data[n] = x;
	}
}

//------ Main Function -----
int main(int argc, char** argv) {
	GpuTimer timer;
	srand(1);
	double init_datastructure = 0.0, fir = 0.0, fill_ds = 0.0, out_ds = 0.0;
	timer.Start();
	int nDevices;
	hipGetDeviceCount(&nDevices);
	int numberOfThreads, numberOfBlocks, maxThreadsPerBlock;
	int totalGlobalMemory = 0;
	for (int i = 0; i < nDevices; i++) {
		hipDeviceProp_t prop;
		hipGetDeviceProperties(&prop, i);
		maxThreadsPerBlock = prop.maxThreadsPerBlock;
	}
	// Stream Initialisation	
	hipStream_t streams[number_of_gpus];
	for (int k = 0; k < number_of_gpus; k++) {
		hipSetDevice(k);
		hipStreamCreate(&streams[k]);
	}

	int h_input_xOffset = 0;
	int h_coeff_xOffset = 0;
	timer.Stop();
	init_datastructure += timer.Elapsed();
	timer.Start();
	// Allocate DataStructures and
	// no values present on host to initialize devices (data will be initialised directly on devices)
	// Allocate Structures
	tmp_size_t = 250013696 * sizeof(float);
	float* d_input0;
	hipSetDevice(0);
	hipMalloc(&d_input0, tmp_size_t);
	float* d_output0;
	hipSetDevice(0);
	hipMalloc(&d_output0, tmp_size_t);
	for (int n = 0; n < (250013696); n++) {
		float x = (rand() / (float)RAND_MAX);
		// printf("\n Write at place %d value %f", n , x);
		h_input[n] = x;
	}

	hipMemcpy(d_input0, &h_input[0], tmp_size_t, hipMemcpyHostToDevice);
	// print << <1, 16 >> > (d_input0);
	//no values present on host to initialize devices (data will be initialised directly on devices)
	//Allocate Structures
	tmp_size_t = 16384 * sizeof(float);
	float* d_coeff0;
	hipSetDevice(0);
	hipMalloc(&d_coeff0, tmp_size_t);
	for (int n = 0; n < (16384); n++) {
		float x = (rand() / (float)RAND_MAX);
		// printf("\n Write at place %d value %f", n , x);
		h_coeff[n] = x;
	}
	hipMemcpyAsync(d_coeff0, &h_coeff[0], tmp_size_t, hipMemcpyHostToDevice, streams[0]);

	int ntaps = 4;
	int nchans = 32768;
	int nspectra = 122071;
	timer.Stop();
	fill_ds += timer.Elapsed();
	timer.Start();

	if (250013696 <= maxThreadsPerBlock) {
		numberOfBlocks = 1;
		numberOfThreads = 250013696;
	}
	else {
		numberOfThreads = maxThreadsPerBlock;
		numberOfBlocks = ceil(250013696.0 / numberOfThreads);
	}
	printf("Working on %d blocks and %d threads!\n", numberOfBlocks, numberOfThreads);

	//MapIndexInPlace Call
	hipSetDevice(0);
	//	int a, int y, int taps, int rowOffset, int* nspectra, int* p_input, int* p_coeff
	FIR_MapIndexInPlaceSkeleton_array << <numberOfBlocks, numberOfThreads, 0, streams[0] >> > (h_input_xOffset + 0, 0, (ntaps), (nchans), (nspectra), d_input0, d_coeff0, d_output0);
	timer.Stop();
	fir += timer.Elapsed();
	timer.Start();
	fetch_size = 250013696 * sizeof(float);
	hipMemcpy(&h_input[0], d_output0, fetch_size, hipMemcpyDeviceToHost);
	//Show h_input as h_input_glbcopy
	//hostdata is up-to-date, just make copy_variablename accessible
	h_input_glbcopy = h_input;

	fetch_size = 16384 * sizeof(float);

	hipMemcpyAsync(&h_coeff[0], d_coeff0, fetch_size, hipMemcpyDeviceToHost, streams[0]);
	h_coeff_glbcopy = h_coeff;

	// Free DataStructures
	hipFree(d_input0);
	hipFree(d_input0_loccopy);
	hipFree(d_input0_glbcopy);
	hipFree(d_coeff0);
	hipFree(d_coeff0_loccopy);
	hipFree(d_coeff0_glbcopy);
	timer.Stop();
	out_ds += timer.Elapsed();
	printf("\n%f; %f; %f; %f", init_datastructure, fill_ds, fir, out_ds);

	// MPI Finalisation
	return EXIT_SUCCESS;
}

