#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include "timer.cuh"
#include ""
#include <stdio.h>
#include <array>
#include <iostream>
#include <time.h>
#include <chrono>
#include <stdlib.h>
#include <stdint.h>
#include <stdio.h>
#include <vector>
#include <hip/hip_runtime.h>
#include <stdexcept>
#include <sstream>

// Global Constants
const size_t number_of_processes = 1;
const size_t number_of_gpus = 1;
// Global Variables
// Temporary Variables
size_t tmp_size_t = 0;
size_t fetch_size = 0;

//------ DataStructure Declarations -----
	//Declare everything for input
std::vector<float> h_input(250118144); //Size:250118144, localSize: 250118144, gpuSize: 250118144
std::vector<float> h_input_glbcopy(250118144);
float* d_input0_loccopy;
float* d_input0_glbcopy;

//Declare everything for coeff
std::vector<float> h_coeff(131072); //Size:480, localSize: 480, gpuSize: 480
std::vector<float> h_coeff_glbcopy(131072);
float* d_coeff0_loccopy;
float* d_coeff0_glbcopy;

//------ Kernel Functions -----
__global__ void print(float* y) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	printf("y at %d:%f\n", index, y[index]);
}
// h_input_xOffset + 0, 0, (ntaps), (nchans), (nspectra), d_input0, d_output0
__global__ void FIR_MapIndexInPlaceSkeleton_array(int a, int y, int taps, int rowOffset, int nspectra, float* p_input, float* p_coeff, float* d_output) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int channels = rowOffset;
	float newa = 0;
	for (int j = 0; ((j) < (taps)); j++) {
		if (((index + ((j) * (channels)))) <= (nspectra * channels)) {
			newa += (// TODO: ExpressionGenerator.generateCollectionElementRef: Array, global indices, distributed
				p_input[(index + ((j) * (channels)))]
				* // TODO: ExpressionGenerator.generateCollectionElementRef: Array, global indices, distributed
				p_coeff[(((j) * (channels)))]
				);
		}
		else {
			// Do nothing since we do not have further data
		}
	}
	d_output[index] = (newa);
}

void writefloattofilevector(std::size_t datasize, std::vector<float> data) {
	for (int n = 0; n < (datasize); n++) {
		float x = (rand() / (float)RAND_MAX);
		// printf("\n Write at place %d value %f", n , x);
		data[n] = x;
	}
}

//------ Main Function -----
int main(int argc, char** argv) {
	GpuTimer timer;
	srand(1);
	double init_datastructure = 0.0, fir = 0.0, fill_ds = 0.0, out_ds = 0.0;
	timer.Start();
	int nDevices;
	hipGetDeviceCount(&nDevices);
	int numberOfThreads, numberOfBlocks, maxThreadsPerBlock;
	int totalGlobalMemory = 0;
	for (int i = 0; i < nDevices; i++) {
		hipDeviceProp_t prop;
		hipGetDeviceProperties(&prop, i);
		maxThreadsPerBlock = prop.maxThreadsPerBlock;
	}
	// Stream Initialisation	
	hipStream_t streams[number_of_gpus];
	for (int k = 0; k < number_of_gpus; k++) {
		hipSetDevice(k);
		hipStreamCreate(&streams[k]);
	}

	int h_input_xOffset = 0;
	int h_coeff_xOffset = 0;
	timer.Stop();
	init_datastructure += timer.Elapsed();
	timer.Start();
	// Allocate DataStructures and
	// no values present on host to initialize devices (data will be initialised directly on devices)
	// Allocate Structures
	tmp_size_t = 250118144 * sizeof(float);
	float* d_input0;
	hipSetDevice(0);
	hipMalloc(&d_input0, tmp_size_t);
	float* d_output0;
	hipSetDevice(0);
	hipMalloc(&d_output0, tmp_size_t);
	for (int n = 0; n < (250118144); n++) {
		float x = (rand() / (float)RAND_MAX);
		// printf("\n Write at place %d value %f", n , x);
		h_input[n] = x;
	}

	hipMemcpy(d_input0, &h_input[0], tmp_size_t, hipMemcpyHostToDevice);
	// print << <1, 16 >> > (d_input0);
	//no values present on host to initialize devices (data will be initialised directly on devices)
	//Allocate Structures
	tmp_size_t = 131072 * sizeof(float);
	float* d_coeff0;
	hipSetDevice(0);
	hipMalloc(&d_coeff0, tmp_size_t);
	for (int n = 0; n < (131072); n++) {
		float x = (rand() / (float)RAND_MAX);
		// printf("\n Write at place %d value %f", n , x);
		h_coeff[n] = x;
	}
	hipMemcpyAsync(d_coeff0, &h_coeff[0], tmp_size_t, hipMemcpyHostToDevice, streams[0]);

	int ntaps = 4;
	int nchans = 32768;
	int nspectra = 7630;
	timer.Stop();
	fill_ds += timer.Elapsed();
	timer.Start();

	if (250118144 <= maxThreadsPerBlock) {
		numberOfBlocks = 1;
		numberOfThreads = 250118144;
	}
	else {
		numberOfThreads = maxThreadsPerBlock;
		numberOfBlocks = ceil(250118144.0 / numberOfThreads);
	}
	printf("Working on %d blocks and %d threads!\n", numberOfBlocks, numberOfThreads);

	//MapIndexInPlace Call
	hipSetDevice(0);
	//	int a, int y, int taps, int rowOffset, int* nspectra, int* p_input, int* p_coeff
	FIR_MapIndexInPlaceSkeleton_array << <numberOfBlocks, numberOfThreads, 0, streams[0] >> > (h_input_xOffset + 0, 0, (ntaps), (nchans), (nspectra), d_input0, d_coeff0, d_output0);
	timer.Stop();
	fir += timer.Elapsed();
	timer.Start();
	fetch_size = 250118144 * sizeof(float);
	hipMemcpy(&h_input[0], d_output0, fetch_size, hipMemcpyDeviceToHost);
	//Show h_input as h_input_glbcopy
	//hostdata is up-to-date, just make copy_variablename accessible
	h_input_glbcopy = h_input;

	fetch_size = 131072 * sizeof(float);

	hipMemcpyAsync(&h_coeff[0], d_coeff0, fetch_size, hipMemcpyDeviceToHost, streams[0]);
	h_coeff_glbcopy = h_coeff;

	// Free DataStructures
	hipFree(d_input0);
	hipFree(d_input0_loccopy);
	hipFree(d_input0_glbcopy);
	hipFree(d_coeff0);
	hipFree(d_coeff0_loccopy);
	hipFree(d_coeff0_glbcopy);
	timer.Stop();
	out_ds += timer.Elapsed();
	printf("\n%f; %f; %f; %f", init_datastructure, fill_ds, fir, out_ds);

	// MPI Finalisation
	return EXIT_SUCCESS;
}

